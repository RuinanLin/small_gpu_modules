#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to perform vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 20; // Size of the vectors (1 million elements)
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(N - i);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify results
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            success = false;
            break;
        }
    }

    std::cout << "Vector addition " << (success ? "PASSED" : "FAILED") << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
